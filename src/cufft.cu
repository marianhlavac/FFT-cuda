/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */
/*https://github.com/drufat/cuda-examples/blob/master/cuda/fft.cu*/

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>

// includes, project
#include <hipfft/hipfft.h>

using namespace std;

// Complex data type
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale(Complex*, const Complex*, int, float);

// Filtering functions
void Convolve(const Complex*, int, const Complex*, int, Complex*);

// Padding functions
int PadData(const Complex*, Complex**, int,
            const Complex*, Complex**, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE        50
#define FILTER_KERNEL_SIZE 11

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    runTest(argc, argv);
    runTest(argc, argv);
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv)
{
  int signal_size = atoi(argv[1]);
    printf("[simpleCUFFT] is starting...\n");

    // Allocate host memory for the signal
    Complex* h_signal = (Complex*)malloc(sizeof(Complex) * signal_size);
    // Initalize the memory for the signal
    for (unsigned int i = 0; i < signal_size; ++i) {
        h_signal[i].x = rand() / (float)RAND_MAX;
        h_signal[i].y = 0;
    }

    // Allocate host memory for the filter
    Complex* h_filter_kernel = (Complex*)malloc(sizeof(Complex) * FILTER_KERNEL_SIZE);
    // Initalize the memory for the filter
    for (unsigned int i = 0; i < FILTER_KERNEL_SIZE; ++i) {
        h_filter_kernel[i].x = rand() / (float)RAND_MAX;
        h_filter_kernel[i].y = 0;
    }

    // Pad signal and filter kernel
    Complex* h_padded_signal;
    Complex* h_padded_filter_kernel;
    int new_size = PadData(h_signal, &h_padded_signal, signal_size,
                           h_filter_kernel, &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
    int mem_size = sizeof(Complex) * new_size;
    cout << new_size << endl;

    // Allocate device memory for signal
    Complex* d_signal;
    hipMalloc((void**)&d_signal, mem_size);
    // Copy host memory to device
    hipMemcpy(d_signal, h_padded_signal, mem_size,
               hipMemcpyHostToDevice);

    // Allocate device memory for filter kernel
    Complex* d_filter_kernel;
    hipMalloc((void**)&d_filter_kernel, mem_size);

    // Copy host memory to device
    hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
               hipMemcpyHostToDevice);
               
    // Start the stopwatch
    auto start = chrono::high_resolution_clock::now();

    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, new_size, HIPFFT_C2R, 1);

    // CUFTT exec
    hipfftExecC2R(plan, (hipfftComplex *)d_signal, (hipfftReal *)d_signal);

    auto finish = chrono::high_resolution_clock::now();    
    
    auto microseconds = chrono::duration_cast<std::chrono::microseconds>(finish-start);

    cout << "elapsed " << microseconds.count() << "us" << endl;
    //Destroy CUFFT context
    hipfftDestroy(plan);

    // cleanup memory
    free(h_signal);
    free(h_filter_kernel);
    free(h_padded_signal);
    free(h_padded_filter_kernel);
    hipFree(d_signal);
    hipFree(d_filter_kernel);

}

// Pad data
int PadData(const Complex* signal, Complex** padded_signal, int signal_size,
            const Complex* filter_kernel, Complex** padded_filter_kernel, int filter_kernel_size)
{
    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;
    int new_size = signal_size + maxRadius;

    // Pad signal
    Complex* new_data = (Complex*)malloc(sizeof(Complex) * new_size);
    memcpy(new_data +           0, signal,              signal_size * sizeof(Complex));
    memset(new_data + signal_size,      0, (new_size - signal_size) * sizeof(Complex));
    *padded_signal = new_data;

    // Pad filter
    new_data = (Complex*)malloc(sizeof(Complex) * new_size);
    memcpy(new_data +                    0, filter_kernel + minRadius,                       maxRadius * sizeof(Complex));
    memset(new_data +            maxRadius,                         0, (new_size - filter_kernel_size) * sizeof(Complex));
    memcpy(new_data + new_size - minRadius,             filter_kernel,                       minRadius * sizeof(Complex));
    *padded_filter_kernel = new_data;

    return new_size;
}

////////////////////////////////////////////////////////////////////////////////
// Filtering operations
////////////////////////////////////////////////////////////////////////////////

// Computes convolution on the host
void Convolve(const Complex* signal, int signal_size,
              const Complex* filter_kernel, int filter_kernel_size,
              Complex* filtered_signal)
{
    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;
    // Loop over output element indices
    for (int i = 0; i < signal_size; ++i) {
        filtered_signal[i].x = filtered_signal[i].y = 0;
        // Loop over convolution indices
        for (int j = - maxRadius + 1; j <= minRadius; ++j) {
            int k = i + j;
            if (k >= 0 && k < signal_size)
                filtered_signal[i] = ComplexAdd(filtered_signal[i], ComplexMul(signal[k], filter_kernel[minRadius - j]));
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b)
{
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(Complex* a, const Complex* b, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
        a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
}